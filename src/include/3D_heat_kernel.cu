#include "hip/hip_runtime.h"
# include <hip/hip_runtime.h>
# include <stdio.h>


__global__ void temperature_update(float& temp1_d, float& temp2_d, float alpha, 
                    float dt, 
                    const int N_x, const int N_y, const int N_z,
                    const float dx, const float dy, const float dz){

    // Load a slice into shared memory:
    __shared__ float slice[blockDim.y + 2][blockDim.x + 2];


    int ix = blockIdx.x*blockDim.x + threadIdx.x;
    int iy = blockIdx.y*blockDim.y + threadIdx.y;
    int in_idx = iy*N_x + ix;  
    int out_idx = 0;
    int stride = dimx*dimy;


    int tx = threadIdx.x + 1;
    int ty = threadIdx.y + 1;

    // The thread has a "personal" copy of the variable
    // in front of and behind the slice:

    float   current = temp1_d[in_idx],
            infront = temp1_d[in_idx + stride],
            behind  = temp1_d[in_idx + 2*stride];

    // Since the "radius" is really just 1:
    for (int i = 1; i < N_z - 1; i++){

        // Advance the slice:
        behind  = current;
        current = infront;
        infront = in_idx;

        in_idx += stride;
        out_idx += stride;

        __syncthreads();

        // Update the data slice in shared mem:
        if (threadIdx.y<1){ // Halo above/below
            slice[threadIdx.y][tx]              = temp1_d[out_idx-N_x];
            slice[threadIdx.y+blockDim.y+1][tx] = temp1_d[out_idx+blockDim.y*N_x];
        }

        if (threadIdx.x<1){ // Halo left/right
            slice[ty][threadIdx.x]              = temp1_d[out_idx-1];
            slice[ty][threadIdx.x+blockDim.x+1] = temp1_d[out_idx+blockDim.x];
        }

        slice[ty][tx] = current;
        __syncthreads();

        // Update temperature at output point:

        temp2_d[out_idx] =  current + 
                            (alpha*dt)*((slice[ty-1][tx] - 
                            2*current + slice[ty+1][tx])/(dx*dx) + 
                            (slice[ty][tx-1] - 
                            2*current + slice[ty][tx+1])/(dy*dy) +
                            (infront - 2*current + behind)/(dz*dz));

    }

}